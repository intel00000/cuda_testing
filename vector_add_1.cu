#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *A, int *B, int *C, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Unique thread index in grid
    if (index < N)
    {                                   // Prevent out-of-bounds memory access
        C[index] = A[index] + B[index]; // Element-wise addition
    }
}

int main()
{
    // reset device
    hipDeviceReset();
    const int N = 4096;       // Number of elements
    int size = N * sizeof(int); // Size in bytes

    // Allocate memory on host (CPU)
    int h_A[N], h_B[N], h_C[N];

    // Initialize vectors with values
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate memory on device (GPU)
    int *d_A, *d_B, *d_C;
    // testing: malloc these memory at CPU
    // d_A = (int *)malloc(size);
    // d_B = (int *)malloc(size);
    // d_C = (int *)malloc(size);

    if (hipMalloc((void **)&d_A, size) != hipSuccess)
    {
        std::cerr << "Error allocating memory for d_A" << std::endl;
        return -1;
    }
    if (hipMalloc((void **)&d_B, size) != hipSuccess)
    {
        std::cerr << "Error allocating memory for d_B" << std::endl;
        return -1;
    }
    if (hipMalloc((void **)&d_C, size) != hipSuccess)
    {
        std::cerr << "Error allocating memory for d_C" << std::endl;
        return -1;
    }
    // Copy input data from host to device
    if (hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cerr << "Error copying data from host to device for d_A" << std::endl;
        return -1;
    }
    if (hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cerr << "Error copying data from host to device for d_B" << std::endl;
        return -1;
    }

    // Define execution configuration
    int threadsPerBlock = 64;                                    // Number of threads per block
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock; // Compute number of blocks

    std::cout << "Number of blocks: " << numBlocks << ", Threads per block: " << threadsPerBlock << std::endl;

    // Launch kernel
    clock_t start, end;
    start = clock();
    vectorAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    if (hipGetLastError() != hipSuccess)
    {
        std::cerr << "Error launching kernel" << std::endl;
        return -1;
    }
    end = clock();
    double time_taken = double(end - start) / CLOCKS_PER_SEC;
    std::cout << "CUDA Time taken: " << time_taken << " seconds" << std::endl;

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print results
    // std::cout << "Vector Addition Results: " << std::endl;
    // for (int i = 0; i < N; i++)
    // {
    //     std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    // }

    // Free device memory
    if (hipFree(d_A) != hipSuccess)
    {
        std::cerr << "Error freeing memory for d_A" << std::endl;
        return -1;
    }
    if (hipFree(d_B) != hipSuccess)
    {
        std::cerr << "Error freeing memory for d_B" << std::endl;
        return -1;
    }
    if (hipFree(d_C) != hipSuccess)
    {
        std::cerr << "Error freeing memory for d_C" << std::endl;
        return -1;
    }
    // Free host memory
    // free(d_A);
    // free(d_B);
    // free(d_C);

    return 0;
}
